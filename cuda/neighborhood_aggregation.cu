#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS_PER_BLOCK 256
#define MAX_NEIGHBORS 64  // Assume K <= MAX_NEIGHBORS for simplicity

// Enhanced CUDA kernel for neighborhood aggregation without warp reduction.
__global__ void enhanced_neighborhood_aggregation_kernel(
    const float* __restrict__ points,    // (B, N, C)
    const int* __restrict__ neighbors,     // (B, N, K)
    float* __restrict__ aggregated,        // (B, N, C)
    int B, int N, int K, int C)             // Dimensions: Batch, Points, Neighbors, Channels
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = B * N;
    if (idx >= total)
        return;

    int b = idx / N;
    int n = idx % N;

    // Each thread computes the aggregation for one point over its K neighbors:
    for (int c = 0; c < C; ++c) {
        float sum = 0.0f;
        #pragma unroll
        for (int k = 0; k < K; ++k) {
            int neighbor_idx = neighbors[b * N * K + n * K + k];
            float val = points[b * N * C + neighbor_idx * C + c];
            sum += val;
        }
        aggregated[b * N * C + n * C + c] = sum / float(K);
    }
}

// Wrapper function exposed to Python.
torch::Tensor enhanced_neighborhood_aggregation(
    torch::Tensor points,    // (B, N, C)
    torch::Tensor neighbors, // (B, N, K)
    int C)                   // Number of channels
{
    auto B = points.size(0);
    auto N = points.size(1);
    auto K = neighbors.size(2);
    auto aggregated = torch::zeros({B, N, C}, points.options());

    int total = B * N;
    int threads = THREADS_PER_BLOCK;
    int blocks = (total + threads - 1) / threads;

    // Launch kernel without dynamic shared memory.
    enhanced_neighborhood_aggregation_kernel<<<blocks, threads>>>(
        points.data_ptr<float>(),
        neighbors.data_ptr<int>(),
        aggregated.data_ptr<float>(),
        B, N, K, C
    );
    hipError_t err = hipGetLastError();
    if (err != hipSuccess){
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    return aggregated;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("enhanced_neighborhood_aggregation", &enhanced_neighborhood_aggregation, "Enhanced Neighborhood Aggregation CUDA Kernel");
}